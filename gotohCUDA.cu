#include <iostream>
#include <fstream> 
#include <vector>
#include <string>
#include <algorithm>
#include <limits>
#include <thread>
#include <mutex>
#include <condition_variable>
#include <queue>
#include <functional>
#include <hip/hip_runtime.h>

using namespace std;

#define IDX(i, j, w) ((i) * (w) + (j))


//this function is a function to read fasta file, i used GPT to write it
string fastaReader(const string &path) {
    ifstream in(path);
    if (!in) throw runtime_error("Cannot open FASTA file: " + path);
    string line, seq;
    while (getline(in, line)) {
        if (line.empty() || line[0] == '>') continue;
        seq += line;
    }
    return seq;
}

//For comparing results 
struct AlignmentResult {
    std::string A_aligned;
    std::string B_aligned;
    int score;

    AlignmentResult(const std::string& a, const std::string& b, int s)
        : A_aligned(a), B_aligned(b), score(s) {}


};

// for transferring variables between function calls
struct AlignmentMatrices {
    std::vector<int> M, I, D;
    std::vector<int> traceM, traceI, traceD;
    int m, n, width;
};



//Computes one anti-diagonal
__global__ void gotohAUX2(int diag, int m, int n, const char* A, const char* B,
                int openGap, int extendGap, const int* submat,
                int* M, int* I, int* D, int*traceM, int* traceI, int*traceD){
            size_t index = blockIdx.x * blockDim.x + threadIdx.x;
            int i = max(1, diag-n) + index;
            int j = diag - i;

            if (i > m || j < 1 || j > n) return;

            int width = n + 1;
            int idx       = IDX(i, j, width);
            int up        = IDX(i - 1, j, width);
            int left      = IDX(i, j - 1, width);
            int diagPrev  = IDX(i - 1, j - 1, width);



            int sub = submat[(int)A[i - 1] * 128 + (int)B[j - 1]];

            // Compute I (insertion score)
            //Here we open new gap vertically
            int openI = M[up] - (openGap + extendGap);
            //Extend existing vertical gap
            int extI  = I[up] - extendGap;
            if (openI >= extI) {
                I[idx] = openI;
                traceI[idx] = 0; 
            } else {
                I[idx] = extI;
                traceI[idx] = 1;
            }

            
            // Compute D (deletion score)
            int openD = M[left] - (openGap + extendGap);
            int extD  = D[left] - extendGap;
            if (openD >= extD) {
                D[idx] = openD;
                traceD[idx] = 0;
            } else {
                D[idx] = extD;
                traceD[idx] = 1;
            }



            // Compute M (match/mismatch score)
            //We have three possibilities here: match/mismatch from M, gap from I or gap from D
            int diagS = M[diagPrev] + sub;

            if (diagS >= I[idx] && diagS >= D[idx]) {
                M[idx] = M[diagPrev] + sub;
                traceM[idx] = 0;

            } else if (I[idx] >= D[idx]) {
                M[idx] = I[idx];
                traceM[idx] = 1;

            } else {
                M[idx] = D[idx];
                traceM[idx] = 2;
            }

    }

AlignmentMatrices gotoch_align_cuda(const string&A, const string &B, int openGap, int extendGap, const std::vector<std::vector<int>> &submat) {
    
    const int THREADS_PER_BLOCK = 128;
    const int NEG_INF = numeric_limits<int>::min()/2;
    
    char *d_A, *d_B;
    int *d_M, *d_I, *d_D, *d_submat;
    int *d_traceM, *d_traceI, *d_traceD;

    

    
    const int m = A.size();
    const int n = B.size();
    const int width = n + 1;
    const int size = (m + 1) * (n + 1);

    //Flatten substition matrix (code received from CHATGPT)
    std::vector<int> flatSubmat(128 * 128, 0);
    for (int i = 0; i < 128; ++i)
        for (int j = 0; j < 128; ++j)
            flatSubmat[i * 128 + j] = submat[i][j];

    
    

    //here we are allocating host flattened matrices (HOST)
    vector<int> hM(size, NEG_INF);
    vector<int> hI(size, NEG_INF);
    vector<int> hD(size, NEG_INF);
    hM[0] = hI[0] = hD[0] = 0;

    for (int i = 1; i <= m; i++) {
        hI[i * width] =  -(openGap +(i-1)*extendGap);
    }
    for (int j = 1; j <= n; j++) {
        hD[j] = -(openGap + (j-1)*extendGap);
    }


    vector<int> hTraceM(size);
    vector<int> hTraceI(size);
    vector<int> hTraceD(size);




    // Allocate to host
    hipMalloc(&d_A, m*sizeof(char));
    hipMalloc(&d_B, n*sizeof(char));
    hipMemcpy(d_A, A.c_str(), m, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.c_str(), n, hipMemcpyHostToDevice);

    hipMalloc(&d_M, size * sizeof(int));
    hipMalloc(&d_I, size * sizeof(int));
    hipMalloc(&d_D, size * sizeof(int));
    hipMemcpy(d_M, hM.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_I, hI.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_D, hD.data(), size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_submat, 128 * 128 * sizeof(int));
    hipMemcpy(d_submat, flatSubmat.data(), 128 * 128 * sizeof(int), hipMemcpyHostToDevice); 


    //traceback matrices
    hipMalloc(&d_traceM, size * sizeof(int));
    hipMalloc(&d_traceI, size * sizeof(int));
    hipMalloc(&d_traceD, size * sizeof(int));

    //Compute on GPU, we launch kernels for each diagonal
    for (int diag = 2; diag <= m + n; ++diag){
        int i_min = max(1, diag - n);
        int i_max = min(m, diag - 1);
        int total = i_max - i_min + 1;
        if (total <= 0) continue;

        int blocks = (total + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
        gotohAUX2<<<blocks, THREADS_PER_BLOCK>>>(diag, m, n, d_A, d_B,
                                         openGap, extendGap, d_submat,
                                         d_M, d_I, d_D,
                                         d_traceM, d_traceI, d_traceD);


        hipDeviceSynchronize();


    }

    //Results back to host
    hipMemcpy(hM.data(), d_M, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hI.data(), d_I, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hD.data(), d_D, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hTraceM.data(), d_traceM, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hTraceI.data(), d_traceI, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hTraceD.data(), d_traceD, size * sizeof(int), hipMemcpyDeviceToHost);


    // compare values of different matrices




    //Free CUDA memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M);
    hipFree(d_I);
    hipFree(d_D);
    hipFree(d_traceM);
    hipFree(d_traceI);
    hipFree(d_traceD);
    hipFree(d_submat);

    return {
        std::move(hM), std::move(hI), std::move(hD),
        std::move(hTraceM), std::move(hTraceI), std::move(hTraceD),
        m, n, width
    };

}

AlignmentResult gotoch_align_result(const std::string& A, const std::string& B,
                                const std::vector<int>& hM,
                                const std::vector<int>& hI,
                                const std::vector<int>& hD,
                                const std::vector<int>& hTraceM,
                                const std::vector<int>& hTraceI,
                                const std::vector<int>& hTraceD,
                                int m, int n, int width) {
    auto IDX = [width](int i, int j) { return i * width + j; };


    int end = IDX(m, n, width);
    int score_D = hD[end];
    int score_M = hM[end];
    int score_I = hI[end];
    int score;
    char chosen;

    if (score_M >= score_I && score_M >= score_D) {
        score = score_M;
        chosen = 'M';
    } else if (score_D >= score_I) {
        score = score_D;
        chosen = 'D';
    }else {
        score = score_I;
        chosen = 'I';
    }
    
    char current = chosen;

    //TO DO: add traceback logic as in CPU
    std::string A_aligned, B_aligned;

    int i = m, j = n;

    while (i > 0 || j > 0) {
        if (current == 'M') {
            if (i == 0) {
                current = 'D';
                continue;
            }
            if (j == 0) {
                current = 'I';
                continue;
            }
            int trace = hTraceM[IDX(i, j, width)];
            if (trace == 0) { // from M[i-1][j-1]
                A_aligned += A[i - 1];
                B_aligned += B[j - 1];
                i--; j--;
            } else if (trace == 1) {
                current = 'I';
            } else {
                current = 'D';
            }
        } else if (current == 'I') {
            if (i == 0) {
                current = 'D';
                continue;
            }
            A_aligned += A[i - 1];
            B_aligned += '-';
            if (hTraceI[IDX(i, j, width)] == 0) {
                current = 'M';
            } else {
                current = 'I';
            }
            i--;
        } else { // current == 'D'
            if (j == 0) {
                current = 'I';
                continue;
            }
            B_aligned += B[j - 1];
            A_aligned += '-';
            if (hTraceD[IDX(i, j, width)] == 0) {
                current = 'M';
            } else {
                current = 'D';
            }
            j--;
        }
    }

    //we reverse so that the strings we get are from left to right 
    std::reverse(A_aligned.begin(), A_aligned.end());
    std::reverse(B_aligned.begin(), B_aligned.end());

    return AlignmentResult(A_aligned, B_aligned, score);
}


int main() {

    //testing code with input strings
    std::string A = fastaReader("seqA.fasta");
    std::string B = fastaReader("seqB.fasta");

    // parameters for scoring alignment
    int openGap = 10;
    int extendGap = 1;
    int match = 3;
    int mismatch = -1;

    // submatrix initialization
    vector<vector<int>> submat(128, vector<int>(128, mismatch));
    for (char c : {'A','C','G','T'}) submat[c][c] = match;

    // get results (body of code)
    AlignmentMatrices ms = gotoch_align_cuda(A, B, openGap, extendGap, submat);
    AlignmentResult result = gotoch_align_result(A, B, ms.M, ms.I, ms.D, ms.traceM, ms.traceI, ms.traceD, ms.m, ms.n, ms.width);

    std::cout << "Aligned A: " << result.A_aligned << "\n";
    std::cout << "Aligned B: " << result.B_aligned << "\n";
    std::cout << "Alignment score: " << result.score << "\n";

    return 0;
}
  

    


  

    

